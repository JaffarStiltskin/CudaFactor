#include "hip/hip_runtime.h"
/*
 * fillEnsemble.cu
 *
 * Created on: 5 Feb. 2014
 * 	  Autor : tony
 */

#include "header/fillEnsemble.h"
#include "header/prime.h"
#include <string.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/**
 * retourne true si l'entier est B-friable false sinon. (Version cpu)
 */
bool isBSmooth(int *list,int size, int y){

	int i =0;
	bool val = true;
	if(list == NULL){
		fprintf(stderr,"list null\n");
		return -1;
	}
	while(i < size){
		if(list[i] > y){
			return false;
		}
		i++;
	}
	return val;
}
/**
 * Retourne true si l'entier fait partie de l'ensemble false sinon
 * (version CPu)
 */
bool isInEnsemble(ensemble ens, int y,int size){
	int i =0;
	if(ens == NULL){
		return false;
	}
	while(i <size ){
		if(ens[i].ind.val == y){
			return true;
		}
		i++;
	}
	return false;
}

/**
 * Mode GPU.
 * Verifie si chaque valeur contenue dans la liste des premiers (de taille size)
 * est inférieur a la valeur y.
 */
__device__ bool isInf(int *list, int size, int y){
	int i = threadIdx.x;
	volatile __shared__ bool res;
	if(i == 0){
		res = true;
	}
	__syncthreads();
	if(i < size){

		if(list[i] > y){
			res=false;
		}
		__syncthreads();
		if(!res){
			return res;
		}
	}
	return res;
}

/**
 * Mode GPU
 * Verifie si la valeur y est B-friable.
 * list est la liste des premiers de la borne
 * result le resultat retourné
 */
__device__ void isBSmoothG(int *list,int size, int y,int *result){
	int i =threadIdx.x;
	volatile __shared__ int found;
	if(threadIdx.x == 0) found = 0;
	__syncthreads();

	if(found== 0 && i < size){
		bool inf = isInf(list,size,y);
		if(inf){
			found = 1;
			*result = found;
		}
		__syncthreads();
		*result = found;
	}
}

/**
 * Mode GPU
 * Verifie si la valeur y fait partie de l'ensemble ens et stocke le resultat
 * res.
 * size est la taille de l'ensemble.
 */

__device__ void isInEnsembleG(ensemble ens, int y,int size, int *res){
	int i =threadIdx.x;
	volatile __shared__ bool found;
	if(i == 0 ){
		found = false;
	}
	__syncthreads();

	if(i <size ){
		if(ens[i].ind.val == y){
			found = true;
			*res=found;
		}
		__syncthreads();
		*res = found;
	}

}


/**
 * Construit l'ensemble R. (version CPU)
 */
void fillEnsemble(ensemble r,int nbr,int borne,ensemble div
		,int sizeDiv){
	int m=0;
	int k;
	int *p = generatePrimeList(borne,&k);
	r = initEns(&m);

	int x;
	int y;
	int racN=sqrt(nbr);

	srand(time(NULL));
	printf("m : %i\n",m);
	while(m <= k+1){
		x = racN+(rand() % ((nbr-1) - racN));
		y = pow(x,2);
		y=y%nbr;
		printf("k : %i\n",isInEnsemble(div,y,sizeDiv));

		printf("m : %i\n",isBSmooth(p,k,y));
		if(isBSmooth(p,k,y) && !isInEnsemble(div,y,sizeDiv)){
			addCouple(r,x,y,&m);

		}
	}
}

__device__ ensemble *generateTab(int borne, int k,int *p,int nbr){
	int i = threadIdx.x;
	int size;

	size = k+1;
	if(i <= size ){
		int x;
		int y;
		int racN=sqrtf(nbr);

		//x = racN+(rand_r(srand(i)) % ((nbr-1) - racN));
		y = powf(x,2);
		y=y%nbr;


	}
	return NULL;
}

__global__ void fillEnsembleG(ensemble r,int *p,int k,int nbr,int borne
		,ensemble div,int sizeDiv,int *sizeR){
	int i = blockIdx.x;
    hiprandState s;
	int m=0;


	int y;
	int racN=sqrtf(nbr);
	int *result;
    hiprand_init(1234+i, i, 0, &s);

    int x = hiprand(&s);
sizeR[i] = x;
    if(i <= k+1){



		//y = powf(x,2);
		//y=y%nbr;
		//isBSmoothG(p,k,y,result);
		//isInEnsembleG(div,y,sizeDiv,result);

	}
}

