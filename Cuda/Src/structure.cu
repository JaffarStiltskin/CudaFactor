/*
 * structure.c
 *
 *  Created on: 6 févr. 2014
 *      Author: tony
 */
#include "header/structure.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
/**
 * Alloue de l'espace memoire pour l'ensemble. Retourne un ensemble vide
 * et initialise size à 0.
 */
ensemble initEns(int *size){
	ensemble tp  = (ensemble) malloc(sizeof(struct cell));
	*size = 0;

	return tp;
}


/**
 * Ajoute (en tete) un couple dans l'ensemble ens et retourne la taille de
 * l'ensemble si tout c'est bien passé sinon retourne NULL. !!ATTENTION POSSIBLE
 * FUITE MEMOIRE!!
 */
__host__ __device__ int addCouple(ensemble ens, int x, int y,int *size){
	*size = *size+1;

	ensemble tp = (ensemble) malloc((*size)*sizeof(struct cell));
	if(*size > 1){

		if(tp == NULL){
			printf("malloc nok size:= %i\n",*size);
			return -1;
		}
		if( memcpy(&tp,&ens,sizeof(ens)) == NULL){
			printf("erreur de recopie d'ensemble\n");
			return -1;
		}

		ens =(ensemble)malloc((*size)*sizeof(struct cell));
		if(ens ==NULL){
			printf("erreur malloc nouvel ensemble\n");
			return -1;
		}

	}

	couple c ;
	c.x = x;
	c.y= y;

	if(memcpy(&tp[*size-1].ind.couple ,&c,sizeof(couple)) == NULL){
		printf("erreur de recopie d'ensemble\n");
		return -1;
	}

	if(memcpy(&ens,&tp,sizeof(tp)) == NULL){
		printf("erreur de recopie d'ensemble\n");
		return -1;
	}
	return 1;
}
/**
 * Ajoute (en tete) une valeur dans l'ensemble ens et retourne la taille de
 * l'ensemble si tout c'est bien passé sinon retourne NULL
 */
__host__ __device__ int addVal(ensemble ens, int x,int *size){
	*size = *size+1;
	ensemble tp = (ensemble) malloc((*size)*sizeof(struct cell));
	if(tp == NULL){
		return -1;
	}
	memcpy(&tp,&ens,sizeof(ens));
	tp[*size-1].ind.val=x;

	return 1;

}
