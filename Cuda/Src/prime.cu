
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void eratosthene(int *list, int borne) {
	int id = blockIdx.x;
	int limite = (int) sqrt((double) borne);
	for (int i = id + 1; i <= borne; i++) {
		if (list[i] != 0 && (list[i] % list[id]) == 0) {
			list[i] = 0;
		}
	}
}

__global__ void listNumbers(int *list) {
	int id = blockIdx.x;
	if (id != 0 && id != 1) {
		list[id - 2] = id;
	}
}

__global__ void copyTab(int *src, int *dest, int size) {
	int id = blockIdx.x;
	if (id < size) {
		dest[id] = src[id];
	}
}

int primeList(int *list, int *result, int borne) {
	int id = 0;
	int res;
	int *dev_res;
	int *dev_list;
	for (int i = 0; i < borne; i++) {
		if (list[i] != 0) {
			result[id] = list[i];
			id++;
		}
	}
	res = hipMalloc(&dev_list, id);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	res = hipMalloc(&dev_res, id);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	res = hipMemcpy(dev_list, result, id, hipMemcpyHostToDevice);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	copyTab<<<id, 1>>>(dev_list, dev_res, id);
	res = hipMemcpy(result, dev_res, id, hipMemcpyDeviceToHost);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	hipFree(dev_list);
	hipFree(dev_res);
	return id;
}

int main(int argc, char **argv) {
	int *numbers;
	int *dev_numbers;
	int borne;
	if (argc == 1) {
		borne = 100;
	} else {
		borne = atoi(argv[1]);
	}
	numbers = (int *) malloc((borne - 1) * sizeof(int));
	int ret = hipMalloc(&dev_numbers, ((borne - 1) * sizeof(int)));
	if (ret != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	//borne + 1 pour inclure la borne
	listNumbers<<<borne + 1, 1>>>(dev_numbers);
	eratosthene<<<borne, 1>>>(dev_numbers, borne);
	hipMemcpy(numbers, dev_numbers, (borne - 1) * sizeof(int), hipMemcpyDeviceToHost);
	if (ret != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	int *result = (int *) malloc(borne * sizeof(int));
	int taille = primeList(numbers, result, borne);
	for (int i = 0; i < taille; i++) {
		printf("%d\n", result[i]);
	}
	hipFree(dev_numbers);
	free(numbers);
	free(result);
	return EXIT_SUCCESS;
}
