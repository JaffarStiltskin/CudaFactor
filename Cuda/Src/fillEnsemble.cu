#include "hip/hip_runtime.h"
/*
 * fillEnsemble.cu
 *
 * Created on: 5 Feb. 2014
 * 	  Autor : tony
 */

#include "header/fillEnsemble.h"
#include "header/prime.h"
#include <string.h>



/**
 * retourne true si l'entier est B-friable false sinon. (Version cpu)
 */
bool isBSmooth(int *list,int size, int y){

	int i =0;
	bool val = true;

	while(i < size){
		if(list[i] > y){
			val = false;
			break;
		}
		i++;
	}
	return val;
}
/**
 * retourne true si l'entier fait partie de l'ensemble false sinon
 * (version CPu)
 */
bool isInEnsemble(ensemble ens, int y,int size){
	int i =0;

	while(i <size ){
		if(ens[i].ind.val == y){
			return true;
		}
		i++;
	}
	return false;
}
__device__ bool isInf(int *list, int size, int base){
	int i = threadIdx.x;
	if(i < size){


		if(list[i] > base){
			return false;
		} else {
			return true;
		}
	}
}
__global__ void isBSmoothh(int *list,int size, int base, int y,int *result){
	printf("blahh\n");
int i =threadIdx.x;
	volatile __shared__ int found;
	if(threadIdx.x == 0) found = 1;
	__syncthreads();

	if(found== 1 && i < size){
		bool inf = isInf(list,size,base);
		if(!inf){
			found = 0;
			*result = found;
		}
		if(threadIdx.x == 0 && *result){
			found = false;
		}
		__syncthreads();
	}
*result=true;
}



__global__ void isInEnsembleG(ensemble ens, int y,int size, int *res){
	int i =0;
printf("kkkk");

	while(i <size ){
		if(ens[i].ind.val == y){
			*res = 1;
			break;
		}
		i++;
	}
	*res = 0;
}


/**
 * Construit l'ensemble R. (version CPU)
 */
void fillEnsemble(ensemble ens,int n,int base,ensemble div,int sizeDiv){
	int m=0;
	int k,sizeE;
	int size;
	ensemble e = initEns(&size);
	int *res = generatePrimeList(base,&k);

	while(m < k+1){
		int r = sqrt(n);
		srand(time(NULL));
		int x = (int)rand()/(double)RAND_MAX * (n-1) + r;
		int y = pow(x,2);
		y=y%n;
		int ysize;
		int *list = generatePrimeList(y,&ysize);
		if(isBSmooth(list,ysize,y) && !isInEnsemble(div,y,sizeDiv)){
			printf("x : %i\n y : %i",x,y);
			if((e=addCouple(ens,x,y,&sizeE)) == NULL){
				fprintf(stderr,"fillEnsemble: erreur lors de l'ajout du couple "
						"dans l'ensemble\n");
				exit(EXIT_FAILURE);
			}
			m = sizeE;
		}
	}
	ens = e;
}

