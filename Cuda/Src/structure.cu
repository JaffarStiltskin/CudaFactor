/*
 * structure.c
 *
 *  Created on: 6 févr. 2014
 *      Author: tony
 */
#include "header/structure.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
/**
 * Alloue de l'espace memoire pour l'ensemble. Retourne un ensemble vide
 * et initialise size à 0.
 */
__host__  __device__ ensemble initEns(int *size){
	ensemble tp  = (ensemble) malloc(sizeof(struct cell));
	*size = 0;

	return tp;
}


/**
 * Ajoute (en tete) un couple dans l'ensemble ens et retourne la taille de
 * l'ensemble si tout c'est bien passé sinon retourne NULL
 */
__host__ __device__ ensemble addCouple(ensemble ens, int x, int y,int *size){
	*size = *size+1;
	ensemble tp = (ensemble) malloc(*size*sizeof(struct cell));
	if(tp == NULL){
			return NULL;
		}
	memcpy(&tp,&ens,sizeof(ens));
	tp[*size-1].ind.couple.x = x;
	tp[*size-1].ind.couple.y = y;

	return tp;

}
/**
 * Ajoute (en tete) une valeur dans l'ensemble ens et retourne la taille de
 * l'ensemble si tout c'est bien passé sinon retourne NULL
 */
__host__ __device__ ensemble addVal(ensemble ens, int x,int *size){
	*size = *size+1;
	ensemble tp = (ensemble) malloc(*size*sizeof(struct cell));
	if(tp == NULL){
			return NULL;
		}
	memcpy(&tp,&ens,sizeof(ens));
	tp[*size-1].ind.val=x;

	return tp;

}
