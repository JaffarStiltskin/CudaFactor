#include "hip/hip_runtime.h"
/*
 *  TestFillEnsemble.c
 *  
 *
 *  Created by Tony on 21/02/2014.
 *  Copyright 2014 __MyCompanyName__. All rights reserved.
 *
 */

#include "TestFillEnsemble.h"
#include "../Src/header/prime.h"
#include "../Src/header/fillEnsemble.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

int TestIsBSmooth(){
	int size;
	int borne = 100;
	int val = 200;
	int *list = generatePrimeList(borne,&size);

	assert(isBSmooth(list, size, val));

	val = 10;
	assert(!isBSmooth(list, size, val));

	return 0;
}
int TestIsInEnsemble(){
	int size,i;
	ensemble e = initEns(&size);

	for (i = 0; i < 32; i++){
		addVal(e,i,&size);
	}
	assert(isInEnsemble(e,12,size) == 1);
	assert(isInEnsemble(e,44,size) == 0);

	return 0;
}
__global__ void isInfKernel(int *dev_list,bool *result,int size,int val){
	int i =threadIdx.x;
	volatile __shared__ bool found;
	if(threadIdx.x == 0) found = false;
	__syncthreads();

	if(found== false  && i < size){
		bool inf = isInf(dev_list,size,val);
		if(inf){
			found = true;
			*result = found;
		}
		__syncthreads();
		*result = found;
	}
}

int TestIsInf(){
	int borne = 99;
	int val = 20;
	int size;
	int *list = generatePrimeList(borne,&size);
	int *dev_list;
	bool *dev_result;
	bool *result=(bool *) malloc(sizeof(bool));

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(bool));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);
	int i;

	isInfKernel<<<1,size>>>(dev_list,dev_result,size,val);
	hipMemcpy(result,dev_result,sizeof(bool),hipMemcpyDeviceToHost);
	assert(*result == true);

	free(result);
	hipFree(dev_result);

	val = 200;
	hipMalloc(&dev_result,sizeof(bool));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	isInfKernel<<<1,size>>>(dev_list,dev_result,size,val);
	hipMemcpy(result,dev_result,sizeof(bool),hipMemcpyDeviceToHost);

	assert(*result == false);

	//free(result);
	free(list);
	hipFree(dev_result);
	hipFree(dev_list);

	return 0;
}
__global__ void IsBSmoothKernel(int *list,int size, int y,int *result){

	isBSmoothG(list,size,y,result);
}
int TestIsBSmoothG(){
	int borne = 99;
	int val = 20;
	int size;
	int *list = generatePrimeList(val,&size);

	int *dev_list;
	int *dev_result;
	int *result=(int *) malloc(sizeof(int));

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	IsBSmoothKernel<<<1,size>>>(dev_list,size,borne,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 1);

	hipFree(dev_result);
	free(result);
	free(list);
	size = 0;
	val = 200;
	list = generatePrimeList(val,&size);

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	result = (int *) malloc(sizeof(int));
	IsBSmoothKernel<<<1,size>>>(dev_list,size,borne,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 0);
	hipFree(dev_result);
	free(result);
	free(list);
	return 0;
}

__global__ void IsInEnsembleKernel(ensemble ens,int size, int y,int *result){

	isInEnsembleG(ens,size,y,result);
	//*result = 1;

}

int TestIsInEnsembleG(){

	int size;
	ensemble ens = initEns(&size);
ensemble dev_ens;
	int val = 22;

	int *dev_result;
	int i;
	for (i = 0; i < 32; i++){
		addVal(ens,i,&size);
	}

	int *result=(int *) malloc(size*sizeof(int));

		hipMalloc(&dev_ens,size*sizeof(struct cell));
		hipMalloc(&dev_result,size*sizeof(int));
		hipMemcpy(dev_ens,ens,size*sizeof(struct cell),hipMemcpyHostToDevice);

		IsInEnsembleKernel<<<1,size>>>(dev_ens,size,val,dev_result);
		hipMemcpy(result,dev_result,size*sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 1);
	free(result);
	hipFree(dev_result);
	val = 20045;
	hipMalloc(&dev_result,sizeof(int));
	int *result2 = (int *) malloc(sizeof(int));
	IsInEnsembleKernel<<<1,size>>>(ens,size,val,dev_result);
	hipMemcpy(result2,dev_result,sizeof(int),hipMemcpyDeviceToHost);
printf("result %i\n",*result2);
	assert(*result == 0);

	return 0;
}


int TestfillEnsemble(){
	int size;
	int nbr = 257;
	int borne = 10;
	ensemble div = initEns(&size);
	ensemble e ;
	fillEnsemble(e,nbr,borne,div,size);
	return 0;
}
int TestfillEnsembleG(){
	int sizediv;
	int *size;
	hipMalloc(&size,5*sizeof(int));
	int k;
	int nbr = 257;
	int borne = 10;
	ensemble div = initEns(&sizediv);
	ensemble e ;
	int *p =generatePrimeList(borne,&k);
	int *x=(int *)malloc(5*sizeof(int));
	fillEnsembleG<<<5,1>>>(e,p,k,nbr,borne,div,sizediv,size);
	hipMemcpy(x,size,5*sizeof(int),hipMemcpyDeviceToHost);
	printf("x :%i\n",x[0]);
	printf("x :%i\n",x[1]);
	printf("x :%i\n",x[2]);
	printf("x :%i\n",x[3]);
	return 0;
}

