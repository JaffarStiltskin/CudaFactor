#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "TestFillEnsemble.h"
#include "TestPgcd.h"
#include "TestStructure.h"


int main(){
	int val = 62;
	int valx = 44;
	int valy = 32;

	testInitEns();
	printf("Test InitEns -> passed\n");

	testAddVal(val);
	printf("Test AddVal -> passed\n");

	testAddCouple(valx,valy);
	printf("Test AddCouple -> passed\n");

	TestIsBSmooth();
	printf("Test BSmooth -> passed\n");

	TestIsInEnsemble();
	printf("Test IsInEnsemble -> passed\n");

	TestIsInf();
	printf("Test isinf -> passed\n");

	TestIsBSmoothG();
	printf("Test BSmoothG -> passed\n");

	TestIsInEnsembleG();
	printf("Test IsInEnsembleG -> passed\n");

	TestfillEnsemble();
	printf("Test fillEnsemble -> passed\n");

	TestfillEnsembleG();
	printf("Test fillEnsembleG -> passed\n");

	TestPgcd();
	printf("Test Pgcd -> passed\n");
	return 0;
}
