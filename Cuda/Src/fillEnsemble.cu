#include "hip/hip_runtime.h"
/*
 * fillEnsemble.cu
 *
 * Created on: 5 Feb. 2014
 * 	  Autor : tony
 */

#include "header/fillEnsemble.h"
#include "header/prime.h"
#include <string.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/**
 * retourne true si l'entier est B-friable false sinon. (Version cpu)
 */
bool isBSmooth(int *list,int size, int y){

	int i =0;
	bool val = true;
	if(list == NULL){
		fprintf(stderr,"list null\n");
		return -1;
	}
	while(i < size){
		if(list[i] > y){
			return false;
		}
		i++;
	}
	return val;
}
/**
 * Retourne true si l'entier fait partie de l'ensemble false sinon
 * (version CPu)
 */
bool isInEnsemble(ensemble ens, int y,int size){
	int i =0;
	if(ens == NULL){
		return false;
	}
	while(i <size ){
		if(ens[i].ind.val == y){
			return true;
		}
		i++;
	}
	return false;
}

/**
 * Mode GPU.
 * Verifie si chaque valeur contenue dans la liste des premiers (de taille size)
 * est inférieur a la valeur y.
 */
__device__ int isInf(int *list, int size, int y){
	int i = threadIdx.x;
	volatile __shared__ int res;
	if(i == 0){
		res = 0;
	}
	__syncthreads();
	if(i < size){

		if(list[i] > y){
			res=1;
		}
		__syncthreads();
	}
	return res;
}

/**
 * Mode GPU
 * Verifie si la valeur y est B-friable.
 * list est la liste des premiers de la borne
 * result le resultat retourné
 */
__device__ void isBSmoothG(int *list,int size, int y,int *result){
	int i =threadIdx.x;
	volatile __shared__ int found;
	if(threadIdx.x == 0) found = 0;
	__syncthreads();

	if(found== 0 && i < size){
		bool inf = isInf(list,size,y);
		if(!inf){
			found = 1;
			*result = found;
		}
		__syncthreads();
		*result = found;
	}
}

/**
 * Mode GPU
 * Verifie si la valeur y fait partie de l'ensemble ens et stocke le resultat
 * res.
 * size est la taille de l'ensemble.
 */

__device__ void isInEnsembleG(ensemble ens, int y,int size, int *res){
	int i = threadIdx.x;
	volatile __shared__ int found;

	if (i == 0){
		found =0;
	}
	if(i < size){
		__syncthreads();
		if(ens[i].ind.val == y){
			found = 1;
			*res = found;
		}
		__syncthreads();
		*res = found;
	}
}

/**
 * Construit l'ensemble R. (version CPU)
 */
void fillEnsemble(ensemble r,int nbr,int borne,ensemble div
		,int sizeDiv){
	int m=0;
	int k;
	int *p = generatePrimeList(borne,&k);
	k--;
	r = initEns(&m);

	int x;
	int y;
	int racN=sqrt(nbr);

	srand(time(NULL));

	while(m < k+1){
		x = racN+(rand() % ((nbr-1) - racN));
		y = pow(x,2);
		y=y%nbr;

		if(isBSmooth(p,k,y) && !isInEnsemble(div,y,sizeDiv)){
			addCouple(&r,x,y,&m);

		}
	}
}

__device__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
	int id = threadIdx.x;
	int racN=sqrtf(seed);

	hiprand_init ( seed+id, id, racN, &state[id] );
}

__device__ void generate( hiprandState* globalState, int *rand)
{
	int ind = threadIdx.x;
	hiprandState localState = globalState[ind];
	*rand =(int) hiprand_uniform( &localState );
	globalState[ind] = localState;
}

/*int generateRandom(int n)
{
	dim3 tpb(n,1,1);
	hiprandState* devStates;
	hipMalloc ( &devStates, N*sizeof( hiprandState ) );

	// setup seeds
	//setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );

	// generate random numbers
	//generate <<< 1, tpb >>> ( devStates );

	return 0;
}*/

__device__ int generateRonce(ensemble r,int *p,int k,int nbr,ensemble div,int sizeDiv, int *sizeR){
	int i = blockIdx.x;

	//TODO ajouter l'aléa
	int y;
	int *x=(int*)malloc(sizeof(int));
	int *bsmooth = (int *)malloc(sizeof(int));
	int *present = (int *)malloc(sizeof(int));

	hiprandState* devStates;
	devStates=(hiprandState*) malloc (k*sizeof( hiprandState ) );
	// setup seeds
	setup_kernel( devStates, nbr );

	// generate random numbers
	generate( devStates, x);

	if(i < k+1){

		y = powf(*x,2);
		y=y%nbr;
		isBSmoothG(p,k,y,bsmooth);
		isInEnsembleG(div,y,sizeDiv,present);

		if((*bsmooth) && !(*present)){
			addCouple(&r,*x,y,sizeR);
			return 0;
		}
	}
	return -1;
}
__global__ void fillEnsembleG(ensemble r,int *p,int k,int nbr,int borne
		,ensemble div,int sizeDiv,int *sizeR){
	int res;
	//TODO la boucle.
	do{
		res = generateRonce(r,p,k,nbr,div,sizeDiv,sizeR);
	}while(res != 0);
}

