#include "hip/hip_runtime.h"
/*
 * fillEnsemble.cu
 *
 * Created on: 5 Feb. 2014
 * 	  Autor : tony
 */

#include "header/fillEnsemble.h"
#include "header/prime.h"
#include <string.h>



/**
 * retourne true si l'entier est B-friable false sinon. (Version cpu)
 */
bool isBSmooth(int *list,int size, int y){

	int i =0;
	bool val = true;
	if(list == NULL){
		fprintf(stderr,"list null\n");
		return -1;
	}
	while(i < size){
		if(list[i] > y){
			val = false;
			break;
		}
		i++;
	}
	return val;
}
/**
 * Retourne true si l'entier fait partie de l'ensemble false sinon
 * (version CPu)
 */
bool isInEnsemble(ensemble ens, int y,int size){
	int i =0;
	if(ens == NULL){
		return false;
	}
	while(i <size ){
		if(ens[i].ind.val == y){
			return true;
		}
		i++;
	}
	return false;
}

/**
 * Mode GPU.
 * Verifie si chaque valeur contenue dans la liste des premiers (de taille size)
 * est inférieur a la valeur y.
 */
__device__ bool isInf(int *list, int size, int y){
	int i = threadIdx.x;
	volatile __shared__ bool res;
	if(i == 0){
		res = true;
	}
	__syncthreads();
	if(i < size){

		if(list[i] > y){
			res=false;
		}
		__syncthreads();
		if(!res){
			return res;
		}
	}
	return res;
}

/**
 * Mode GPU
 * Verifie si la valeur y est B-friable.
 * list est la liste des premiers de la borne
 * result le resultat retourné
 */
__global__ void isBSmoothG(int *list,int size, int y,int *result){
	int i =threadIdx.x;
	volatile __shared__ int found;
	if(threadIdx.x == 0) found = 0;
	__syncthreads();

	if(found== 0 && i < size){
		bool inf = isInf(list,size,y);
		if(inf){
			found = 1;
			*result = found;
		}
		__syncthreads();
		*result = found;
	}
}

/**
 * Mode GPU
 * Verifie si la valeur y fait partie de l'ensemble ens et stocke le resultat
 * res.
 * size est la taille de l'ensemble.
 */

__global__ void isInEnsembleG(ensemble ens, int y,int size, int *res){
	int i =threadIdx.x;
	volatile __shared__ bool found;
	if(i == 0 ){
		found = false;
	}
	__syncthreads();

	if(i <size ){
		if(ens[i].ind.val == y){
			found = true;
			*res=found;
		}
		__syncthreads();
		*res = found;
	}

}


/**
 * Construit l'ensemble R. (version CPU)
 */
void fillEnsemble(ensemble r,int nbr,int borne,ensemble div,int sizeDiv){
	int m=0;
	int k;
	int *p = generatePrimeList(borne,&k);
	r = initEns(&m);

	int x;
	int y;
	int racN=sqrt(nbr);

	srand(time(NULL));

	while(m <= k+1){
		x = racN+(rand() % ((nbr-1) - racN));
		y = pow(x,2);
		y=y%nbr;

		if(isBSmooth(p,k,y) && !isInEnsemble(div,y,sizeDiv)){
			addCouple(r,x,y,&m);

		}
	}
}

