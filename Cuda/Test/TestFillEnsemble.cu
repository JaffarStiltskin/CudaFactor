#include "hip/hip_runtime.h"
/*
 *  TestFillEnsemble.c
 *  
 *
 *  Created by Tony on 21/02/2014.
 *  Copyright 2014 __MyCompanyName__. All rights reserved.
 *
 */

#include "TestFillEnsemble.h"
#include "../Src/header/prime.h"
#include "../Src/header/fillEnsemble.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

int TestIsBSmooth(){
	int size;
	int borne = 100;
	int val = 200;
	int *list = generatePrimeList(borne,&size);
	size--;
	assert(isBSmooth(list, size, val));

	val = 10;
	assert(!isBSmooth(list, size, val));
	free(list);
	return 0;
}
int TestIsInEnsemble(){
	int size,i;
	ensemble e = initEns(&size);

	for (i = 0; i < 32; i++){
		addVal(e,i,&size);
	}
	assert(isInEnsemble(e,12,size) == 1);
	assert(isInEnsemble(e,44,size) == 0);
free(e);
	return 0;
}
__global__ void isInfKernel(int *dev_list,bool *result,int size,int val){
	int i =threadIdx.x;
	volatile __shared__ bool found;
	if(threadIdx.x == 0) found = false;
	__syncthreads();
	//Attention size prend une valeur de trop et bien superieur à la borne !!
	//TODO Rectifier la fonction generatePrimeList
	if(found== false  && i < size-1){
		int inf = isInf(dev_list,size-1,val);
		if(inf){
			found = true;
			*result = found;
		}
		__syncthreads();
		*result = found;
	}
}

int TestIsInf(){
	int borne = 20;
	int val = 8;
	int size;

	int *list = generatePrimeList(borne,&size);
	int *dev_list;
	bool *dev_result;
	bool *result=(bool *) malloc(sizeof(bool));

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(bool));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);


	isInfKernel<<<1,size>>>(dev_list,dev_result,size,val);
	hipMemcpy(result,dev_result,sizeof(bool),hipMemcpyDeviceToHost);
	assert(*result == true);

	free(result);
	hipFree(dev_result);

	val = 200;
	hipMalloc(&dev_result,sizeof(bool));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	isInfKernel<<<1,size>>>(dev_list,dev_result,size,val);
	hipMemcpy(result,dev_result,sizeof(bool),hipMemcpyDeviceToHost);
	assert(*result == false);


	hipFree(dev_result);
	hipFree(dev_list);
	free(result);
	free(list);
	return 0;
}
__global__ void IsBSmoothKernel(int *list,int size, int y,int *result){

	isBSmoothG(list,size,y,result);
}
int TestIsBSmoothG(){
	int borne = 99;
	int val = 20;
	int size;
	int *list = generatePrimeList(val,&size);
	//Rectification temporaire
	size--;
	int *dev_list;
	int *dev_result;
	int *result=(int *) malloc(sizeof(int));

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	IsBSmoothKernel<<<1,size>>>(dev_list,size,borne,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 1);

	hipFree(dev_list);
	hipFree(dev_result);
	free(result);
	free(list);
	size = 0;
	val = 200;
	list = generatePrimeList(val,&size);
	//Rectification temporaire
		size--;
	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	result = (int *) malloc(sizeof(int));
	IsBSmoothKernel<<<1,size>>>(dev_list,size,borne,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 0);

	hipFree(dev_list);
	hipFree(dev_result);
	free(result);
	free(list);
	return 0;
}

__global__ void IsInEnsembleKernel(ensemble ens,int size, int y,int *result){

	isInEnsembleG(ens,y,size,result);


}

int TestIsInEnsembleG(){

	int size;
	ensemble ens = initEns(&size);
	ensemble dev_ens;
	int val = 22;

	int *dev_result;
	int i;
	for (i = 0; i < 32; i++){
		addVal(ens,i,&size);
	}
	printf("size %i\n",size);
	int *result=(int *) malloc(size*sizeof(int));
	printf("size : %i\n",size);
	hipMalloc(&dev_ens,sizeof(struct cell));
	printf("size3 %i\n",size);
	hipMalloc(&dev_result,size*sizeof(int));
	printf("size4 %i\n",size);
	hipMemcpy(dev_ens,ens,size*sizeof(struct cell),hipMemcpyHostToDevice);
	printf("val : %i\n",val);
	IsInEnsembleKernel<<<1,size>>>(dev_ens,size,val,dev_result);
	hipMemcpy(result,dev_result,size*sizeof(int),hipMemcpyDeviceToHost);
	printf("%i\n",*result);
	assert(*result == 1);


	free(result);
	hipFree(dev_result);
	val = 20045;
	result=(int *) malloc(size*sizeof(int));

	hipMalloc(&dev_result,size*sizeof(int));
	printf("val : %i\n",val);
	IsInEnsembleKernel<<<1,size>>>(dev_ens,size,val,dev_result);
	hipMemcpy(result,dev_result,size*sizeof(int),hipMemcpyDeviceToHost);
	printf("%i\n",*result);
	assert(*result == 0);


	hipFree(dev_result);
	free(ens);
	hipFree(dev_ens);

	return 0;
}


int TestfillEnsemble(){
	int size;
	int nbr = 257;
	int borne = 10;
	ensemble div = initEns(&size);
	ensemble e ;
	fillEnsemble(e,nbr,borne,div,size);
	return 0;
}
int TestfillEnsembleG(){
	int sizediv;
	int *size;
	hipMalloc(&size,5*sizeof(int));
	int k;
	int nbr = 257;
	int borne = 10;
	ensemble div = initEns(&sizediv);
	ensemble e ;
	int *p =generatePrimeList(borne,&k);
	int *x=(int *)malloc(5*sizeof(int));
	fillEnsembleG<<<5,1>>>(e,p,k,nbr,borne,div,sizediv,size);
	hipMemcpy(x,size,5*sizeof(int),hipMemcpyDeviceToHost);
	printf("x :%i\n",x[0]);
	printf("x :%i\n",x[1]);
	printf("x :%i\n",x[2]);
	printf("x :%i\n",x[3]);
	return 0;
}

